#include "hip/hip_runtime.h"
inline void gpuAssert(hipError_t code, char const *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#define GEC(ans) { gpuAssert((ans), __FILE__, __LINE__); } // GPU Error Check


// Kernels
__global__ void empiezo(GANA::Tetrahedron *in_IA, int const n) {

	int ti = threadIdx.x + blockIdx.x * blockDim.x;

	if (ti < n) {
        in_IA[ti]._p[0][0] += 10.;
        in_IA[ti]._p[0][1] += 10.;
        in_IA[ti]._p[0][2] += 10.;
        in_IA[ti]._p[1][0] += 10.;
        in_IA[ti]._p[1][1] += 10.;
        in_IA[ti]._p[1][2] += 10.;
        in_IA[ti]._p[2][0] += 10.;
        in_IA[ti]._p[2][1] += 10.;
        in_IA[ti]._p[2][2] += 10.;
        in_IA[ti]._p[3][0] += 10.;
        in_IA[ti]._p[3][1] += 10.;
        in_IA[ti]._p[3][2] += 10.;
	}

	return;
}

__global__ void init_grilla(float* grilla, int grilla_size, float x_min,
	float y_min, float z_min, float x_max, float y_max, float z_max,
	int x_cnt, int y_cnt, int z_cnt, float resolution) {

	const int nproc = gridDim.x * blockDim.x;
	int ti = threadIdx.x + blockIdx.x * blockDim.x;

	while (ti < grilla_size) {
		if (ti % 3 == 0) {
			float x_step = x_min + resolution * ((ti % (x_cnt * 3)) / 3);
			grilla[ti] = x_step;
		} else if (ti % 3 == 1) {
			float y_step = y_min
					+ resolution * ((ti % (x_cnt * y_cnt * 3)) / (3 * x_cnt));
			grilla[ti] = y_step;

		} else { // ti % 3 == 2
			float z_step = z_min + resolution * (ti / (3 * x_cnt * y_cnt));
			grilla[ti] = z_step;
		}
		ti = ti + nproc;
	}

	return;
}


__global__ void in_bbox(float* molecule_points, int x_min, int y_min, int z_min,
                int x_max, int y_max, int z_max, int natoms, bool* d_atoms_in_x,
                bool* d_atoms_in_y, bool* d_atoms_in_z, bool* atoms_in_bbox) {

        int ti = threadIdx.x + blockIdx.x * blockDim.x;

        // Get the atoms that lie inside the planes delimited by [xmin, xmax],
        // [ymin, ymax] and [zmin, zmax].
        if (ti < natoms) {
                d_atoms_in_x[ti] = ( (molecule_points[ti*3] > x_min) &&
                                (molecule_points[ti*3] < x_max) ) ? true : false;

                d_atoms_in_y[ti] = ( (molecule_points[ti*3 + 1] > y_min) &&
                                                (molecule_points[ti*3 + 1] < y_max) ) ? true : false;

                d_atoms_in_z[ti] = ( (molecule_points[ti*3 + 2] > z_min) &&
                                                (molecule_points[ti*3 + 2] < z_max) ) ? true : false;
        }
        __syncthreads();
        // Now get the joint set of the previos atoms to get those that lie inside
        // the cube
        if (ti < natoms) {
                atoms_in_bbox[ti] = d_atoms_in_x[ti] && d_atoms_in_y[ti] &&
                                d_atoms_in_z[ti];
        }

        // Now, index the "molecule_points" array to get those atoms
        return;
}
