float rsltion = .1;
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <stdexcept>
#include <string>
#include <typeinfo>

#include "GANA/utils.hpp"
#include "GANA/continuous.hpp"
#include "GANA/grid.hpp"
#include "GANA/kernels.cu"

int main(int argc, char **argv) {
	// Get positions and associated variables ready.
	if (argc != 4) {
		std::cerr << "Usage: GANA in_pdb resolution ou_pdb" << '\n';
		return 0;
	}

	std::vector<int> indices = {300, 600, 900, 1200, 1500, 1800, 1240,
		400, 500, 700, 800, 1000, 1100};
	///////////

	try {
		rsltion = std::stof(argv[1]);
	} catch(...) {
		std::cerr << "Bad input resolution. Please specify a "
			<< "number between .01 and 1" << '\n';
	}

	/////////////////////////
	GANA::Molecule prote(argv[2]), *Dprote;
	// Paso molécula a GPU. Falta pasar los arrays.
	GEC( hipMalloc((void **) &Dprote, sizeof(GANA::Molecule)) );
	GEC( hipMemcpyAsync(Dprote, &prote, sizeof(GANA::Molecule), hipMemcpyHostToDevice) );

	// Paso los arrays.
	float *Dradii, *Din_radii;
	GANA::Point *Dxyz, *Din_xyz;
	const auto xyz_sz = sizeof(float) * prote._natoms * 3,
		rad_sz = sizeof(float) * prote._natoms;
	GEC( hipMalloc((void **) &Dxyz, xyz_sz) );
	GEC( hipMalloc((void **) &Din_xyz, xyz_sz) );
	GEC( hipMalloc((void **) &Dradii, rad_sz) );
	GEC( hipMalloc((void **) &Din_radii, rad_sz) );
	GEC( hipMemcpyAsync(Dxyz, prote._xyz, xyz_sz, hipMemcpyHostToDevice) );
	GEC( hipMemcpyAsync(Din_xyz, prote._in_xyz, xyz_sz, hipMemcpyHostToDevice) );
	GEC( hipMemcpyAsync(Dradii, prote._radii, rad_sz, hipMemcpyHostToDevice) );
	GEC( hipMemcpyAsync(Din_radii, prote._in_radii, rad_sz, hipMemcpyHostToDevice) );

	// Apunto los pointers de la molécula (en GPU) a los arrays (en GPU).
	GEC( hipMemcpyAsync(&(Dprote->_xyz), &Dxyz, sizeof(GANA::Point *),
		hipMemcpyHostToDevice) );
	GEC( hipMemcpyAsync(&(Dprote->_in_xyz), &Din_xyz, sizeof(GANA::Point *),
		hipMemcpyHostToDevice) );
	GEC( hipMemcpyAsync(&(Dprote->_radii), &Dradii, sizeof(float *),
		hipMemcpyHostToDevice) );
	GEC( hipMemcpyAsync(&(Dprote->_in_radii), &Din_radii, sizeof(float *),
		hipMemcpyHostToDevice) );
	/////////////////////////

	GANA::Triangulation incl_area(prote, indices);
	incl_area.draw("aux/ia.pdb");

//	float *lista_xyz;
//	lista_ptos = (void**) malloc(lista_ptos, sizeof(f))
//	incl_area._Dtetrahedrons

	const dim3 dB0(1024, 1, 1);
	const dim3 dG0(10, 1, 1);
	empiezo<<<dG0, dB0>>>(incl_area._Dtetrahedrons, incl_area._ntetrahedrons);
	hipDeviceSynchronize();
	hipMemcpy(incl_area._tetrahedrons, incl_area._Dtetrahedrons,
			sizeof(GANA::Tetrahedron) * incl_area._ntetrahedrons, hipMemcpyDeviceToHost);

	incl_area.draw("aux/after.pdb");


	return 0;
}
